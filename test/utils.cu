#include "hip/hip_runtime.h"
#include <glog/logging.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <memory>
#include "utils.cuh"

__global__ void test_function_cu(float* cu_arr, int32_t size, float value) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid >= size) {
    return;
  }
  cu_arr[tid] = value;
}

void test_function(float* arr, int32_t size, float value) {
  if (!arr) {
    return;
  }
  float* cu_arr = nullptr;
  hipError_t err = hipMalloc(&cu_arr, sizeof(float) * size);
  CHECK_EQ(err, hipSuccess);  // 检查 hipMalloc 错误

  test_function_cu<<<1, size>>>(cu_arr, size, value);
  hipDeviceSynchronize();
  err = hipGetLastError();
  CHECK_EQ(err, hipSuccess);

  err = hipMemcpy(arr, cu_arr, size * sizeof(float), hipMemcpyDeviceToHost);
  CHECK_EQ(err, hipSuccess);  // 检查 hipMemcpy 错误
  
  hipFree(cu_arr);
}

void set_value_cu(float* arr_cu, int32_t size, float value) {
  int32_t threads_num = 512;
  int32_t block_num = (size + threads_num - 1) / threads_num;
  
  test_function_cu<<<block_num, threads_num>>>(arr_cu, size, value);
  hipDeviceSynchronize();
  const hipError_t err = hipGetLastError();
  CHECK_EQ(err, hipSuccess);
}

__global__ void setup_kernel(hiprandState* states, unsigned int seed, int32_t size) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < size) {
    hiprand_init(seed, tid, 0, &states[tid]);
  }
}

__global__ void generate_random_data_kernel(float* data, int32_t size, hiprandState* states) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid >= size) {
    return;
  }
  
  // Generate random number using cuRAND
  float random = hiprand_uniform(&states[tid]);
  data[tid] = random * 10.0f; // Generate random number between 0 and 10
}

void generate_random_data_cu(float* arr_cu, int32_t size, unsigned int seed) {
  int32_t threads_num = 512;
  int32_t block_num = (size + threads_num - 1) / threads_num;

  // Allocate hiprandState for each thread
  hiprandState* dev_states = nullptr;
  hipError_t err = hipMalloc(&dev_states, size * sizeof(hiprandState));
  CHECK_EQ(err, hipSuccess);

  // 修改 lambda 的写法，接受一个参数
  auto cleanup = [](hiprandState* ptr) {
    if (ptr) {
      hipFree(ptr);
    }
  };
  // 使用 dev_states 作为管理的指针
  std::unique_ptr<hiprandState, decltype(cleanup)> cleanup_guard(dev_states, cleanup);

  // Initialize random number generator states
  setup_kernel<<<block_num, threads_num>>>(dev_states, seed, size);
  err = hipGetLastError();
  CHECK_EQ(err, hipSuccess);
  
  // Generate random numbers
  generate_random_data_kernel<<<block_num, threads_num>>>(arr_cu, size, dev_states);
  hipDeviceSynchronize();
  err = hipGetLastError();
  CHECK_EQ(err, hipSuccess);
}
